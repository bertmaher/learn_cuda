
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <algorithm>
#include <cmath>

__global__
void mish(int n, float* tx, float* aten_mul) {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < n; i += gridDim.x * blockDim.x) {
    float tx_1 = __ldg(tx + i);
    aten_mul[i] = tx_1 * tanhf(log1pf(expf(tx_1)));
  }
}
template<typename T, typename U>
constexpr T ceildiv(T t, U u) {
  return (t + u - 1) / u;
}

int main() {
  constexpr int N = 1 << 28;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N * sizeof(float));
  y = (float*)malloc(N * sizeof(float));
  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 3.0f;
    y[i] = 2.0f;
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  // constexpr int blockSize = 512;
  // constexpr int nBlocks = ceildiv(N, blockSize);
  constexpr int blockSize = 512;
  constexpr int nBlocks = ceildiv(N, blockSize);
  float millis = 0.0f;
  float temp = 0.0f;
  for (int i = 0; i < 500; i++) {
    hipEventRecord(start);
    mish<<<nBlocks, blockSize>>>(N, d_x, d_y);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&temp, start, stop);
    millis += temp;
  }
  millis = millis / 500;

  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
  
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    float mv = 3.0f * tanhf(std::log1p(std::exp(3.0)));
    maxError = std::max(maxError, std::abs(mv - y[i]));
  }
  printf("max error: %f\n", maxError);
  printf("duration (ms): %f\n", millis);
  printf("effective bandwidth (gb/s): %f\n", (float)N * sizeof(float) * 3 / millis / 1e6);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

  return 0;
}

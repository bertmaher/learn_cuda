
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <algorithm>
#include <cmath>

__global__
void mish_gridstride(int n, float* tx, float* aten_mul) {
  for (int i = (threadIdx.x + blockDim.x * blockIdx.x) * 4; i < n; i += gridDim.x * blockDim.x * 4) {
    float4 tx4 = __ldg(reinterpret_cast<float4*>(tx + i));

    tx4.x = tx4.x * tanh(log1p(exp(tx4.x)));
    tx4.y = tx4.y * tanh(log1p(exp(tx4.y)));
    tx4.z = tx4.z * tanh(log1p(exp(tx4.z)));
    tx4.w = tx4.w * tanh(log1p(exp(tx4.w)));

    *reinterpret_cast<float4*>(aten_mul + i) = tx4;
  }
}

__global__
void mish_threadper(int n, float* tx, float* aten_mul) {
  int i = (threadIdx.x  + blockDim.x * blockIdx.x) * 4;
  if (i < n) {
    float4 tx4 = __ldg(reinterpret_cast<float4*>(tx + i));

    tx4.x = tx4.x * tanh(log1p(exp(tx4.x)));
    tx4.y = tx4.y * tanh(log1p(exp(tx4.y)));
    tx4.z = tx4.z * tanh(log1p(exp(tx4.z)));
    tx4.w = tx4.w * tanh(log1p(exp(tx4.w)));

    *reinterpret_cast<float4*>(aten_mul + i) = tx4;
  }
}

__global__
void mish_threadper_fix(int n, float* tx, float* aten_mul) {
  int i = threadIdx.x  + blockDim.x * blockIdx.x;
  if (i < (n / 4)) {
    float4 tx4 = __ldg(reinterpret_cast<float4*>(tx) + i);

    tx4.x = tx4.x * tanh(log1p(exp(tx4.x)));
    tx4.y = tx4.y * tanh(log1p(exp(tx4.y)));
    tx4.z = tx4.z * tanh(log1p(exp(tx4.z)));
    tx4.w = tx4.w * tanh(log1p(exp(tx4.w)));

    reinterpret_cast<float4*>(aten_mul)[i] = tx4;
  }
  int rem = n % 4;
  if (i == n / 4 && rem) {
    while (rem) {
      int idx = n - rem--;
      float elt = tx[idx];
      aten_mul[idx] = elt * tanh(log1p(exp(elt)));
    }
  }
}

template<typename T, typename U>
constexpr T ceildiv(T t, U u) {
  return (t + u - 1) / u;
}

int main() {
  constexpr int N = (1 << 28) + 3;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N * sizeof(float));
  y = (float*)malloc(N * sizeof(float));
  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 3.0f;
    y[i] = 2.0f;
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  {
    constexpr int blockSize = 512;
    constexpr int maxBlocks = 0; //ceildiv(N / 4, blockSize);
    for (int numBlocks = 512; numBlocks <= maxBlocks; numBlocks <<= 1) {
      std::cout << "numBlocks: " << numBlocks << "\n";
      float millis = 0.0f;
      float temp = 0.0f;
      for (int i = 0; i < 500; i++) {
	hipEventRecord(start);
	mish_gridstride<<<numBlocks, blockSize>>>(N, d_x, d_y);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&temp, start, stop);
	millis += temp;
      }
      millis = millis / 500;

      hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
  
      float maxError = 0.0f;
      for (int i = 0; i < N; i++) {
	float mv = 3.0f * tanhf(std::log1p(std::exp(3.0)));
	maxError = std::max(maxError, std::abs(mv - y[i]));
      }
      printf("max error: %f\n", maxError);
      printf("duration (ms): %f\n", millis);
      printf("effective bandwidth (gb/s): %f\n", (float)N * sizeof(float) * 3 / millis / 1e6);
    }
  }
    
  
  for (int algo = 2; algo < 3; algo++) {
    switch (algo) {
    case 0:
      std::cout << "algorithm: grid stride loop\n";
      break;
    case 1:
      std::cout << "algorithm: thread per element\n";
      break;
    case 2:
      std::cout << "algorithm: thread per element with vector tail\n";
      break;
    }
    constexpr int blockSize = 512;
    int nBlocks = ceildiv(N, blockSize) / 4;
    if (algo == 0) {
      nBlocks = 8192;
    }
    float millis = 0.0f;
    float temp = 0.0f;
    for (int i = 0; i < 500; i++) {
      hipEventRecord(start);
      switch (algo) {
      case 0:
	mish_gridstride<<<nBlocks, blockSize>>>(N, d_x, d_y);
	break;
      case 1:
	mish_threadper<<<nBlocks, blockSize>>>(N, d_x, d_y);
	break;
      case 2:
	mish_threadper_fix<<<nBlocks, blockSize>>>(N, d_x, d_y);
	break;
      }	
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&temp, start, stop);
      millis += temp;
    }
    millis = millis / 500;

    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
  
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
      float mv = 3.0f * tanhf(std::log1p(std::exp(3.0)));
      maxError = std::max(maxError, std::abs(mv - y[i]));
    }
    printf("max error: %f\n", maxError);
    printf("duration (ms): %f\n", millis);
    printf("effective bandwidth (gb/s): %f\n", (float)N * sizeof(float) * 2 / millis / 1e6);
  }
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

  return 0;
}


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cmath>
#include <stdio.h>

template<typename T, typename U>
constexpr T ceildiv(T t, U u) {
  return (t + u - 1) / u;
}

__global__
void saxpy(int n, float a, float* x, float* y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = a * x[i] + y[i];
  }
}

int main() {
  constexpr int N = 1 << 20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N * sizeof(float));
  y = (float*)malloc(N * sizeof(float));
  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  constexpr int blockSize = 256;
  constexpr int nBlocks = ceildiv(N, blockSize);
  saxpy<<<nBlocks, blockSize>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = std::max(maxError, std::abs(y[i] - 4.0f));
  }
  printf("max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

  return 0;
}

  
